#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"
#include "stdint.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "timers.h"

#define N (1000000000)

#define THREADS_PER_BLOCK (512)
#define BLOCKS_PER_GRID (131072)


__global__
void graveler(size_t n, uint32_t *one_counts, hiprandState *rand_states, uint64_t seed) {
  int i;
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int nthreads = gridDim.x * blockDim.x;
  hiprandState *state = rand_states + tid;
  hiprand_init(seed + tid, tid, 0, state);

  for(i = tid; i < N; i += nthreads) {
    uint32_t one_count = 0;
    for(size_t rolls = 0; rolls < 231; rolls++) {
      uint32_t roll = (uint32_t)(hiprand_uniform(state) * 3.0 + 0.5f);
      if(roll == 0) {
        ++one_count;
      }
    }
    one_counts[i] = one_count;
  }
}

int main(void) {
  uint32_t *h_one_counts, *d_one_counts;
  hiprandState *d_rand_states;
  hipMalloc(&d_rand_states, sizeof(hiprandState) * THREADS_PER_BLOCK * BLOCKS_PER_GRID);
  hipMalloc(&d_one_counts, N * sizeof(uint32_t));
  h_one_counts = (uint32_t *)malloc(N * sizeof(uint32_t));

  DECLARE_TIMER(GravelerKernelTimer);
  START_TIMER(GravelerKernelTimer);
  graveler<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(N, d_one_counts, d_rand_states, time(0));
  hipDeviceSynchronize();
  STOP_TIMER(GravelerKernelTimer);

  hipMemcpy(h_one_counts, d_one_counts, N * sizeof(uint32_t), hipMemcpyDeviceToHost);

  uint32_t max_ones = 0;
  for(size_t i = 0; i < N; ++i) {
    if(h_one_counts[i] > max_ones) {
      max_ones = h_one_counts[i];
    }
  }
  fprintf(stdout, "Max ones rolled: %u\n", max_ones);
  PRINT_TIMER(GravelerKernelTimer);

  hipFree(d_one_counts);
  hipFree(d_rand_states);
  free(h_one_counts);
  
  return 0;
}