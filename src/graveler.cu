#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"
#include "stdint.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "timers.h"

#define N (1000000000)

#define THREADS_PER_BLOCK (512)
#define BLOCKS_PER_GRID (131072)

__global__ 
void init_device_state(size_t n, hiprandState *rand_states, uint64_t seed) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState *rand_state = rand_states + index;
  hiprand_init(seed, index, 0, rand_state);
}

__global__
void graveler_streaks(size_t n, uint32_t *one_counts, hiprandState *rand_states) {
  size_t i;
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_threads = gridDim.x * blockDim.x;
  hiprandState *rand_state = rand_states + index;

  for(i = index; i < N; i += num_threads) {
    uint32_t one_count = 0;
    for(size_t rolls = 0; rolls < 231; rolls++) {
      uint32_t roll = (uint32_t)(hiprand_uniform(rand_state) * 3.0 + 0.5f);
      if(roll != 0) break;
      ++one_count;
    }
    one_counts[i] = one_count;
  }
}

__global__
void graveler_total(size_t n, uint32_t *one_counts, hiprandState *rand_states) {
  size_t i;
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  size_t num_threads = gridDim.x * blockDim.x;
  hiprandState *rand_state = rand_states + index;

  for(i = index; i < N; i += num_threads) {
    uint32_t one_count = 0;
    for(size_t rolls = 0; rolls < 231; rolls++) {
      uint32_t roll = (uint32_t)(hiprand_uniform(rand_state) * 3.0 + 0.5f);
      if(roll == 0) {
        ++one_count;
      }
    }
    one_counts[i] = one_count;
  }
}

int main(void) {
  uint32_t *h_one_counts, *d_one_counts;
  hiprandState *d_rand_states;
  hipMalloc(&d_rand_states, sizeof(hiprandState) * THREADS_PER_BLOCK * BLOCKS_PER_GRID);
  hipMalloc(&d_one_counts, N * sizeof(uint32_t));
  h_one_counts = (uint32_t *)malloc(N * sizeof(uint32_t));

  int default_device;
  hipDeviceProp_t device_props;
  hipGetDevice(&default_device);
  hipGetDeviceProperties(&device_props, default_device);
  fprintf(stdout, "Using device %s\n", device_props.name);

  init_device_state<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(N, d_rand_states, time(0));
  hipDeviceSynchronize();
  puts("Device initialized");

  DECLARE_TIMER(GravelerKernelTimer);
  START_TIMER(GravelerKernelTimer);
  graveler_streaks<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(N, d_one_counts, d_rand_states);
  hipDeviceSynchronize();
  STOP_TIMER(GravelerKernelTimer);

  hipMemcpy(h_one_counts, d_one_counts, N * sizeof(uint32_t), hipMemcpyDeviceToHost);
  puts("Computation complete");
  puts("Calculating statistics");
  uint32_t max_ones = 0;
  uint32_t min_ones = ~0;
  uint64_t total_ones = 0;
  for(size_t i = 0; i < N; ++i) {
    total_ones += h_one_counts[i];
    if(h_one_counts[i] > max_ones) {
      max_ones = h_one_counts[i];
    }
    if(h_one_counts[i] < min_ones) {
      min_ones = h_one_counts[i];
    }
  }
  fprintf(stdout, "\nNumber of attempts: %lu\nMax ones rolled: %u\nMin ones rolled:%u\nAverage ones: %0.2lf\n",
    N, max_ones, min_ones, (double)total_ones / (double)N);
  PRINT_TIMER(GravelerKernelTimer);

  hipFree(d_one_counts);
  hipFree(d_rand_states);
  free(h_one_counts);
  
  return 0;
}
